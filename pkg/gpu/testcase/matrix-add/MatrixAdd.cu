
#include <hip/hip_runtime.h>
#include <stdio.h>

// 定义矩阵的维度
#define WIDTH 16
#define HEIGHT 16

// CUDA内核用于矩阵加法
__global__ void matrixAdd(const float* A, const float* B, float* C, int width, int height)
{
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    if (col < width && row < height)
    {
        int index = row * width + col;
        C[index] = A[index] + B[index];
    }
}

int main()
{
    int numElements = WIDTH * HEIGHT;
    size_t size = numElements * sizeof(float);

    // 为每个矩阵分配主机内存
    float* h_A = (float*)malloc(size);
    float* h_B = (float*)malloc(size);
    float* h_C = (float*)malloc(size);

    // 初始化矩阵数据
    for (int i = 0; i < numElements; ++i)
    {
        h_A[i] = rand()/(float)RAND_MAX;
        h_B[i] = rand()/(float)RAND_MAX;
    }

    // 为每个矩阵分配设备内存
    float* d_A = NULL;
    hipMalloc((void**)&d_A, size);
    float* d_B = NULL;
    hipMalloc((void**)&d_B, size);
    float* d_C = NULL;
    hipMalloc((void**)&d_C, size);

    // 将矩阵数据从主机复制到设备
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // 定义线程块的大小和网格的大小
    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((WIDTH + threadsPerBlock.x - 1) / threadsPerBlock.x, (HEIGHT + threadsPerBlock.y - 1) / threadsPerBlock.y);
    
    // 启动内核
    matrixAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, WIDTH, HEIGHT);

    // 从设备复制结果回主机
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    // 打印矩阵A
    printf("Matrix A:\n");
    for (int i = 0; i < HEIGHT; ++i)
    {
        for (int j = 0; j < WIDTH; ++j)
        {
            printf("%f ", h_A[i * WIDTH + j]);
        }
        printf("\n");
    }

    // 打印矩阵B
    printf("Matrix B:\n");
    for (int i = 0; i < HEIGHT; ++i)
    {
        for (int j = 0; j < WIDTH; ++j)
        {
            printf("%f ", h_B[i * WIDTH + j]);
        }
        printf("\n");
    }

    // 打印矩阵C
    printf("Matrix C:\n");
    for (int i = 0; i < HEIGHT; ++i)
    {
        for (int j = 0; j < WIDTH; ++j)
        {
            printf("%f ", h_C[i * WIDTH + j]);
        }
        printf("\n");
    }

    // 释放设备内存
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    // 释放主机内存
    free(h_A);
    free(h_B);
    free(h_C);

    return 0;
}