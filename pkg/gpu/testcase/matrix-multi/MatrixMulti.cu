
#include <hip/hip_runtime.h>
#include <stdio.h>

#define WIDTH 16 // 定义矩阵的宽度

// CUDA内核函数，用于矩阵乘法
__global__ void matrixMultiply(const float* A, const float* B, float* C, int width)
{
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    if (row < width && col < width)
    {
        float result = 0.0f;
        for (int k = 0; k < width; ++k)
        {
            result += A[row * width + k] * B[k * width + col];
        }
        C[row * width + col] = result;
    }
}

int main()
{
    int numElements = WIDTH * WIDTH;
    size_t size = numElements * sizeof(float);

    // 分配主机内存
    float* h_A = (float*)malloc(size);
    float* h_B = (float*)malloc(size);
    float* h_C = (float*)malloc(size);

    // 初始化矩阵A和B
    for (int i = 0; i < numElements; ++i)
    {
        h_A[i] = rand()/(float)RAND_MAX;
        h_B[i] = rand()/(float)RAND_MAX;
    }

    // 分配设备内存
    float* d_A = NULL;
    float* d_B = NULL;
    float* d_C = NULL;
    hipMalloc((void**)&d_A, size);
    hipMalloc((void**)&d_B, size);
    hipMalloc((void**)&d_C, size);

    // 将矩阵数据从主机复制到设备
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // 定义线程块和网格大小
    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((WIDTH + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (WIDTH + threadsPerBlock.y - 1) / threadsPerBlock.y);

    // 启动内核计算矩阵乘法
    matrixMultiply<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, WIDTH);

    // 将结果从设备复制回主机
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    // 打印一些结果进行检查
    printf("Matrix A:\n");
    for (int i = 0; i < WIDTH; ++i)
    {
        for (int j = 0; j < WIDTH; ++j)
        {
            printf("%f ", h_A[i * WIDTH + j]);
        }
        printf("\n");
    }

    printf("Matrix B:\n");
    for (int i = 0; i < WIDTH; ++i)
    {
        for (int j = 0; j < WIDTH; ++j)
        {
            printf("%f ", h_B[i * WIDTH + j]);
        }
        printf("\n");
    }

    printf("Matrix C:\n");
    for (int i = 0; i < WIDTH; ++i)
    {
        for (int j = 0; j < WIDTH; ++j)
        {
            printf("%f ", h_C[i * WIDTH + j]);
        }
        printf("\n");
    }

    // 释放设备内存
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    // 释放主机内奇
    free(h_A);
    free(h_B);
    free(h_C);

    return 0;
}